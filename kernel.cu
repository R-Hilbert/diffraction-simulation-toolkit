
#include <hip/hip_runtime.h>
#include<hip/hip_complex.h>

#ifdef __cplusplus
extern"C"{
#endif

typedef struct vector{double x,y,z;}vector;

void kernelCuda(size_t, size_t, vector*, void*, double, double, vector, double, double);
int getDeviceCountCuda();
void setDeviceCuda(int);
void freeCuda(void*);
int getMaxThreadsPerBlockCuda();
void*mallocCuda(size_t);
void memsetCuda(void*, int, size_t);
void memcpyHostToDeviceCuda(void*, const void*, size_t);
void memcpyDeviceToHostCuda(void*, const void*, size_t);

#ifdef __cplusplus
}
#endif

__device__  const hipDoubleComplex i = { 0, 1 };

__global__ void kernel(vector* input, hipDoubleComplex* output, hipDoubleComplex factor, vector avg, double wave, double lambda){
	const int id = blockDim.x*blockIdx.x + threadIdx.x;
	vector in = input[id];
	const double distance = sqrt((in.x - avg.x)*(in.x - avg.x) + (in.y - avg.y)*(in.y - avg.y) + (in.z - avg.z)*(in.z - avg.z));
	hipDoubleComplex exponential;
	sincos(wave*distance,&exponential.y,&exponential.x);
	output[id] = hipCsub(output[id], hipCmul(hipCdiv(i, make_hipDoubleComplex(lambda*distance, 0)), hipCmul(factor, exponential)));
}
void kernelCuda(size_t grid_size,size_t block_size,vector*input,void*output,double real,double imag,vector avg,double wave,double lambda){
	kernel<<<grid_size,block_size>>>(input,(hipDoubleComplex*)output,make_hipDoubleComplex(real,imag),avg,wave,lambda);
}
int getDeviceCountCuda(){
	int i;
	hipGetDeviceCount(&i);
	return i;
}
void setDeviceCuda(int i) {
	hipSetDevice(i);
}
void freeCuda(void*v) {
	hipFree(v);
}
int getMaxThreadsPerBlockCuda() {
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.maxThreadsPerBlock;
}
void*mallocCuda(size_t size) {
	void*v;
	hipMalloc(&v,size);
	return v;
}
void memsetCuda(void*ptr, int value, size_t size) {
	hipMemset(ptr,value,size);
}
void memcpyHostToDeviceCuda(void*dst,const void*src,size_t count) {
	hipMemcpy(dst,src,count,hipMemcpyHostToDevice);
}
void memcpyDeviceToHostCuda(void*dst,const void*src,size_t count) {
	hipMemcpy(dst,src,count,hipMemcpyDeviceToHost);
}